#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <time.h>
#include <cmath>
#include <limits.h>

#define MAXHOPS 4
#define MAX_WAITING_TIME 420
#define BLOCK_LENGTH 512
#define END_OF_ARRAY 2147483647
#define BUFFER_LENGTH 50
#define AIRPORT_PATH "C:/Users/acer/Desktop/Semester 7/Project/AA_airports.txt" //"C:/Users/acer/Desktop/Semester 7/Project/Data/AA_airports.txt"

#define FLIGHT_PATH "C:/Users/acer/Desktop/Semester 7/Project/AA_data1.txt" //"C:/Users/acer/Desktop/Semester 7/Project/Data/OAGX_data_num_1.txt"

bool bool1 = true;
bool bool2 = false;
using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)


int ADJ_MATRIX_DIM;
__device__ int DEV_ADJ_MATRIX_DIM;

// FYP_BFS.cpp : Defines the entry point for the console application.
//

///////////////////Global Variables///////////////////
struct Flight{
	int flightNumber;
	int source;
	int destination;
	int arrivalTime;
	int departureTime;
	int price;
	string code;
};

vector<string> Airport_List;
vector<Flight> Flight_List;
vector<int>** AdjMatrix;

//////////////////////////////////////////////////////

//////////////////Data Read///////////////////////////
int readAirports(){
	
	ifstream myFile;
	myFile.open(AIRPORT_PATH);
	int numberOfAirports=0;
	if(myFile.is_open()){

		
		string line;

		cout<<"Reading Airports"<<endl;
		
		while(myFile.good()){
//------------------------------changed-------------------//			
			//myFile.ignore(256,' ');
			string s;
			myFile>>s;
			Airport_List.push_back(s);
			
			numberOfAirports++;
		}
	}
	myFile.close();
	ADJ_MATRIX_DIM = Airport_List.size();

	//hipMemcpy(DEV_ADJ_MATRIX_DIM,&ADJ_MATRIX_DIM,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(DEV_ADJ_MATRIX_DIM),&ADJ_MATRIX_DIM,sizeof(int),0,hipMemcpyHostToDevice);
	cudaCheckErrors("Error copying adj matrix dim to device");
	cout<<Airport_List.size()<<" Airports Found"<<endl;

	return 1;
}


void readFlights(){

	//this is a bloody array of pointers
	AdjMatrix = new vector<int>*[Airport_List.size()];

	for(int i=0;i<Airport_List.size();i++){
		//thisi is a bloody array of vectors
		AdjMatrix[i] = new vector<int>[Airport_List.size()];
	}

	ifstream myFile;
	myFile.open(FLIGHT_PATH);

	int numOfFlights = 0;
	if(myFile.is_open()){
				
		string line;
		
		Flight tempFlight;
		while(myFile.good()){
//---------------------------------------changed----------------------------------------//			
			tempFlight.flightNumber= numOfFlights;
			/*myFile>>tempFlight.source;
			myFile>>tempFlight.destination;
			myFile>>tempFlight.departureTime;
			myFile>>tempFlight.arrivalTime;
			if(tempFlight.arrivalTime<tempFlight.departureTime) tempFlight.arrivalTime+=10080;
			myFile>>tempFlight.price;
			
			myFile>>tempFlight.code;*/

			myFile>>tempFlight.source;
			myFile>>tempFlight.destination;
			myFile>>tempFlight.price;
			myFile>>tempFlight.departureTime;
			myFile>>tempFlight.arrivalTime;
			if(tempFlight.arrivalTime<tempFlight.departureTime) tempFlight.arrivalTime+=10080;			
			
			myFile>>tempFlight.code;
						
			//add this flight to the adjmatrix;
			Flight_List.push_back(tempFlight);

			AdjMatrix[tempFlight.source][tempFlight.destination].push_back(tempFlight.flightNumber);

			numOfFlights++;

			if(numOfFlights%10000==0) cout<<"*";
		}
			
			cout<<endl;
	}
	myFile.close();
	
	cout<<Flight_List.size()<<" Flights Found"<<endl;
}

/////////////////////////////////////////////////////////////////////////////////////


struct route{
	vector<int> flights;
	int weight;
};



int initializeFlightListInDevice(Flight* &dev_flight_list){
	//allocate space for the flight list in cuda
	hipMalloc((void**)&dev_flight_list, Flight_List.size()*sizeof(Flight));
	cudaCheckErrors("Failed to allocate memory to flight list");	
	hipMemcpy(dev_flight_list,&Flight_List[0],Flight_List.size()*sizeof(Flight),hipMemcpyHostToDevice);
	cudaCheckErrors("Failed to copy flight list");
	
	return 1;
}



int initializeAdjMatrixInDevice(int** &dev_adj_list, int ** &host_adj_vector){

	
	size_t size = ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int*);

	

	//the vector in host that records the pointers in device memory
	host_adj_vector = (int **)malloc(size);
	
	//i indicates rows and j indicates columns of the adjacency matrix
	//allocate device memory for the boolean vector
	
	//allocate memory for each manhattan in device and store the pointer in memory
	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		for(int j=0;j<ADJ_MATRIX_DIM;j++){
			
			hipMalloc((void **)&host_adj_vector[i*ADJ_MATRIX_DIM+j],AdjMatrix[i][j].size()*sizeof(int));
			cudaCheckErrors("Failed to allocate memory to airport list manhattan:");
			hipMemcpy(host_adj_vector[i*ADJ_MATRIX_DIM+j],&AdjMatrix[i][j][0],AdjMatrix[i][j].size()*sizeof(int),hipMemcpyHostToDevice);
			cudaCheckErrors("Failed to copy data to airport list manhattan:");
			
		}
		if(i%100==0) cout<<"&";
	}	
	cout<<endl;

	hipMalloc((void***)&dev_adj_list,size);
	cudaCheckErrors("Failed to allocate memory to pointer list in device");
	hipMemcpy(dev_adj_list,host_adj_vector,size,hipMemcpyHostToDevice);
	cudaCheckErrors("Failed to allocate data to pointer list in device");

	return 1;
}

int initializeBooleanMatrixInDevice(int* &boolean_matrix){
	int* host_bool_matrix= new int[ADJ_MATRIX_DIM*ADJ_MATRIX_DIM];
	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		for(int j=0;j<ADJ_MATRIX_DIM;j++){
			host_bool_matrix[i*ADJ_MATRIX_DIM+j] =  (AdjMatrix[i][j].size() !=0);
		}
	}
	size_t size_bool =ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int);
	hipMalloc((void**)&boolean_matrix,size_bool);
	cudaCheckErrors("Failed to allocate memory to boolean adj matrix");
	hipMemcpy(boolean_matrix,host_bool_matrix,size_bool,hipMemcpyHostToDevice);
	cudaCheckErrors("Failed to move data to boolean adj matrix");
	delete(host_bool_matrix);

	return 1;
}

int initializeBuffer(int* &buffer){
	int* host_bool_buffer= new int[ADJ_MATRIX_DIM*ADJ_MATRIX_DIM];
	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		for(int j=0;j<ADJ_MATRIX_DIM;j++){
			host_bool_buffer[i*ADJ_MATRIX_DIM+j] = false;
		}
	}
	size_t size_bool =ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int);
	hipMalloc((void**)&buffer,size_bool);
	cudaCheckErrors("Failed to allocate memory to boolean buffer");
	hipMemcpy(buffer,host_bool_buffer,size_bool,hipMemcpyHostToDevice);
	cudaCheckErrors("Failed to move data to boolean buffer");
	delete(host_bool_buffer);
	return 1;
}

__global__ void testBuffer(int* buffer,int* result, int size){
	int id =blockIdx.x*blockDim.x+threadIdx.x;
	if(id<size){
		if(buffer[id])
		result[id] = 1234345;
		else
			result[id] = 0;
	}
}

__global__ void testMatrix(int** devVector,int size, int* result, Flight* flights){
	//block dimension is the number of threads in a block. since blockid is zero based multiplying gets you somewhre close. 
	//to gt the correct position all u have to do then is to add the thread id
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	result[i] = 0;
	if(i<size*size && devVector[i]!= NULL )		
		result[i] = flights[devVector[i][0]].source;
}

__global__ void testMatrixBoolean(int* devMatrixBoolean,int size, int* result){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	result[i] = 0;
	//put 1 if a manhattan exists for the particular position
	if(i<size*size && devMatrixBoolean[i])		
		result[i] = 1;
}
//initialize buffer to end of array value so that as values are filled the array size will change, but will still be
//indicated by the first end of array value

//__global__ void initializeBuffer(bool* buffer, int size){
//	int id = blockIdx.x*blockDim.x+threadIdx.x;
//	if(id<size)
//		buffer[id] = false;
//}

//give enough threads to span the source row
//maximum id should be adj_matrix_dimension

__global__ void firstExpansion(int* buffer, int*dev_boolean_matrix, int source){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	//the source row. 
	
	//if(id<DEV_ADJ_MATRIX_DIM*DEV_ADJ_MATRIX_DIM){
	//	//if(dev_adj_matrix[DEV_ADJ_MATRIX_DIM*(source-1)+id]!=NULL){
	//	//	//set source to the precedant node list of each relevant airport
	//	//	buffer[id*BUFFER_LENGTH] = source;
	//	//}
	//}
}

//max id is number of airports
__global__ void expansion(int* dev_buffer,int* boolean_matrix, int* dev_source_vector,int matrix_dimension){
	
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int row = (int) floor((double)id/matrix_dimension);
	int column = id%matrix_dimension;
	if(row<matrix_dimension && column<matrix_dimension){
		
		//for the source row if the matrix row column position has a manhattan set the buffer position to true
		dev_buffer[id] = (dev_source_vector[row] && boolean_matrix[id]);
		
		
	}
	__syncthreads();
	//set the source vector positions to zero by the first of each row
	if(row<matrix_dimension && column<matrix_dimension&& column==0) dev_source_vector[row]= 0;
	__syncthreads();
	
	if((row<matrix_dimension && column<matrix_dimension) && boolean_matrix[id]){
		dev_source_vector[column] = 1;
	}
}

//__global__ void copyNextSource(bool* next_source_array, bool* current_array, int size){
//	int id = blockDim.x*blockIdx.x+threadIdx.x;
//	if(id<size)
//		hipMemcpy
//}

int main(int argc)
{
	
	readAirports();
	readFlights();
	
	int source = 344;
	int destination = 190;

	
	Flight* dev_flight_list;
	
	int** dev_adj_list;
	int* dev_adj_matrix_boolean;
	int** host_adj_vector;
	int* dev_level1;
	int* dev_level2;
	int* frames;
	//boolean array containing source airports in the next expansion
	int* dev_next_source_array;
	
	size_t matrixSize = ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int);
	size_t bufferSize = ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int);
		
	//add the flight array to GPU memory
	cout<<"Initializing Flights"<<endl;	
	initializeFlightListInDevice(dev_flight_list);
	cout<<"finished initializing FLights"<<endl;

	//add the adjacency matrix with manhattans to GPU
	cout<<"Initializing Matrix"<<endl;
	initializeAdjMatrixInDevice(dev_adj_list,host_adj_vector);	
	cout<<"Finished with adj matrix"<<endl;

	//add the boolean adjacency matrix (without manhattans) to GPU
	cout<<"Initializing Boolean Matrix"<<endl;
	initializeBooleanMatrixInDevice(dev_adj_matrix_boolean);	
	cout<<"Finished with boolean matrix"<<endl;

	//allocate memory for the 'next source array' in device
	hipMalloc((void**)&dev_next_source_array,ADJ_MATRIX_DIM*sizeof(int));
	cudaCheckErrors("Failed to allocate memory to next source list");

	int* source_vector = new int [ADJ_MATRIX_DIM];

	//initialize the 'next source vector' with the source row of the adjacency matrix
	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		source_vector[i] = AdjMatrix[source][i].size()!=0;
	}
	//intialize 'next source array' in device
	hipMemcpy(dev_next_source_array,source_vector,ADJ_MATRIX_DIM*sizeof(int),hipMemcpyHostToDevice);
	cudaCheckErrors("Failed to move data  to next source list");

	delete(source_vector);
	//////////////////////initialize the buffers for all the levels/////////////////
	cout<<"initializing Buffers"<<endl;

	initializeBuffer(dev_level1);
	initializeBuffer(dev_level2);
	
	cout<<"initialized buffers"<<endl;
	///////////////////////////////////////Interations///////////////////////////////////////
	
	int numBlocks = ceil((double)ADJ_MATRIX_DIM*ADJ_MATRIX_DIM/BLOCK_LENGTH);

	ofstream myFile;
	myFile.open("nextSource.txt");

	
	int* myArray = (int*) malloc(ADJ_MATRIX_DIM*sizeof(int));
	hipMemcpy(myArray,dev_next_source_array,ADJ_MATRIX_DIM*sizeof(int),hipMemcpyDeviceToHost);
	cudaCheckErrors("Failed to copy data from buffer array to host array");


	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		//if(myArray[i]!= NULL)
			myFile<<myArray[i];
	}
	myFile<<endl<<endl;
	free(myArray);
	cout<<"moving into first expansion"<<endl;

	ofstream myFile2;
	myFile2.open("Frame1.txt");
	int* myArray2 = (int*)malloc(ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int));

	   expansion<<<numBlocks,BLOCK_LENGTH>>>(dev_level1,dev_adj_matrix_boolean,dev_next_source_array,ADJ_MATRIX_DIM);
	 hipDeviceSynchronize();
	 cudaCheckErrors("Error occured in expansion");
	cout<<"finished expansion"<<endl;
	  hipMemcpy(myArray2,dev_level1,ADJ_MATRIX_DIM*ADJ_MATRIX_DIM*sizeof(int),hipMemcpyDeviceToHost);
	 cudaCheckErrors("Failed to retrieve memory from first frame");
	for(int i=0;i<ADJ_MATRIX_DIM*ADJ_MATRIX_DIM;i++){
		//if(myArray[i]!= NULL)
			myFile2<<myArray2[i];
	}

	myFile2<<endl<<endl;
	myFile2.close();

	free(myArray2);
	int* myArray1 = (int*)malloc(ADJ_MATRIX_DIM*sizeof(int));

	/* expansion<<<numBlocks,BLOCK_LENGTH>>>(dev_level1,dev_adj_matrix_boolean,dev_next_source_array,ADJ_MATRIX_DIM);
	 hipDeviceSynchronize();
	 cudaCheckErrors("Error occured in expansion");
	cout<<"finished expansion"<<endl;*/
	 hipMemcpy(myArray1,dev_next_source_array,ADJ_MATRIX_DIM*sizeof(int),hipMemcpyDeviceToHost);
	 cudaCheckErrors("Failed to retrieve memory from buffer array 1.2 to host");
	for(int i=0;i<ADJ_MATRIX_DIM;i++){
		//if(myArray[i]!= NULL)
			myFile<<myArray1[i];
	}
	myFile<<endl<<endl;
	free(myArray1);
	myFile.close();
	
	
	//hipFree(dev_next_source_array);
	
	
	hipFree(dev_level1);
	hipFree(dev_level2);
	hipFree(dev_flight_list);
	
	//for(int i=0;i<ADJ_MATRIX_DIM*ADJ_MATRIX_DIM;i++){
	//	//cout<<i<<endl;
	//	if(host_adj_vector[i] !=NULL)
	//		hipFree(host_adj_vector[i]);
	//}
	hipFree(dev_adj_list);
	free(host_adj_vector);
	
	
	
	return 0;
}

